#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {

    if (a.size() != static_cast<size_t>(n * n) ||
        b.size() != static_cast<size_t>(n * n)) {
        throw std::invalid_argument("not n*n");
    }

    if ((n & (n - 1)) != 0) {
        throw std::invalid_argument("power of 2");
    }

    std::vector<float> result(n * n);
    if (n == 0) return result;

    size_t memory_size = n * n * sizeof(float);
    float *device_a, *device_b, *device_c;

    hipMalloc(&device_a, memory_size);
    hipMalloc(&device_b, memory_size);
    hipMalloc(&device_c, memory_size);

    hipMemcpy(device_a, a.data(), memory_size, hipMemcpyHostToDevice);
    hipMemcpy(device_b, b.data(), memory_size, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha_val = 1.0f;
    const float beta_val = 0.0f;

    hipblasSgemm(handle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_N,
                n,
                n,
                n,
                &alpha_val,
                device_b, n,
                device_a, n,
                &beta_val,
                device_c, n);

    hipMemcpy(result.data(), device_c, memory_size, hipMemcpyDeviceToHost);

    hipblasDestroy(handle);
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    return result;
}